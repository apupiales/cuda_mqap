#include "hip/hip_runtime.h"
/*
 * kernel.cu
 *
 *  Created on: May 19, 2019
 *      Author: Andr�s Pupiales Ar�valo
 *      apupiales@gmail.com
 *      https://github.com/apupiales
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation; either version 2 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * The GNU General Public License is available at:
 *   http://www.gnu.org/copyleft/gpl.html
 * or by writing to:
 *     The Free Software Foundation, Inc.,
 *     675 Mass Ave, Cambridge, MA 02139, USA.
 */

 // System includes
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>
#include <stdbool.h>
#include <math.h>

// CUDA runtime
#include "hip/hip_runtime.h"
#include ""
#include "device_atomic_functions.h"
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"
#include "hipfft/hipfft.h"

//Custom sources
#include "settings_KC10_2fl_1uni.cu"
//#include "settings_KC10_2fl_1uni.cu"
//#include "settings_KC10_2fl_2uni.cu"
//#include "settings_KC10_2fl_3uni.cu"
//#include "settings_KC20_2fl_1uni.cu"
//#include "settings_KC20_2fl_2uni.cu"
//#include "settings_KC20_2fl_3uni.cu"
//#include "settings_KC30_2fl_1uni.cu"
//#include "settings_KC30_2fl_2uni.cu"

// In NSGA2, the Rt population size.
const int NSGA2_POPULATION_SIZE = POPULATION_SIZE * 2;

__global__ void curand_setup(hiprandState* state, int seed) {

	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	hiprand_init(seed, idx, 0, &state[idx]);
}

/**
 * This function creates the base population with POPULATION_SIZE chromosomes
 */
__global__ void generateBasePopulation(short population[][FACILITIES_LOCATIONS]) {

	if (threadIdx.x < FACILITIES_LOCATIONS) {
		population[blockIdx.x][threadIdx.x] = threadIdx.x;
	}
}

/**
 * This function shuffles chromosomes genes randomly over all population
 */
__global__ void shufflePopulationGenes(hiprandState* my_curandstate,
	const unsigned* max_rand_int, const unsigned* min_rand_int,
	short population[][FACILITIES_LOCATIONS]) {

	#pragma unroll
	for (int j = 0; j < FACILITIES_LOCATIONS; j++) {
		int idx = j + blockDim.x * blockIdx.x;

		float myrandf = hiprand_uniform(my_curandstate + idx);
		int myrand = int(myrandf * 10);

		if (myrand != population[blockIdx.x][j]) {
			short current_value = population[blockIdx.x][j];
			population[blockIdx.x][j] = population[blockIdx.x][myrand];
			population[blockIdx.x][myrand] = current_value;
		}
	}
}

/**
 * This function convert the base population to its binary 2d matrix representation.
 * population_2d variable will have the original binary 2d matrix representation
 * of each chromosome and population_2d_transposed will have the transposed version
 * of each binary 2d matrix representation.
 */
__global__ void populationTo2DRepresentation(
	short population[][FACILITIES_LOCATIONS],
	short population_2d[][FACILITIES_LOCATIONS],
	short population_2d_transposed[][FACILITIES_LOCATIONS]) {

	int i = blockIdx.x;
	int j = threadIdx.x;
	int k = threadIdx.y;

	if (j < FACILITIES_LOCATIONS && k < FACILITIES_LOCATIONS) {
		population_2d[j + (i * FACILITIES_LOCATIONS)][k] = 0;
		population_2d_transposed[k + (i * FACILITIES_LOCATIONS)][j] = 0;
		if (population[i][j] == k) {
			population_2d[j + (i * FACILITIES_LOCATIONS)][k] = 1;
			population_2d_transposed[k + (i * FACILITIES_LOCATIONS)][j] = 1;
		}

	}
}

/**
 *  Multiplication between selected flow matrix and input_matrix
 *  (in this strict order)
 */
__global__ void multiplicationWithFlowMatrix(int flow_matrix_id,
	short input_matrix[][FACILITIES_LOCATIONS],
	unsigned int output_matrix[][FACILITIES_LOCATIONS]) {

	int i = blockIdx.x;
	int j = threadIdx.x;
	int k = threadIdx.y;

	if (j < FACILITIES_LOCATIONS && k < FACILITIES_LOCATIONS) {
		unsigned int sum = 0;
		#pragma unroll
		for (int x = 0; x < FACILITIES_LOCATIONS; x++) {
			sum += d_flowMatrices[flow_matrix_id][j][x]
				* input_matrix[x + (i * FACILITIES_LOCATIONS)][k];
		}
		output_matrix[j + (i * FACILITIES_LOCATIONS)][k] = sum;
	}
}

/**
 *  Multiplication between Transposed distance matrix and input_matrix
 *  (in this strict order)
 */
__global__ void multiplicationWithTranposedDistanceMatrix(
	unsigned int input_matrix[][FACILITIES_LOCATIONS],
	unsigned int output_matrix[][FACILITIES_LOCATIONS]) {

	int i = blockIdx.x;
	int j = threadIdx.x;
	int k = threadIdx.y;

	if (j < FACILITIES_LOCATIONS && k < FACILITIES_LOCATIONS) {
		int sum = 0;
		#pragma unroll
		for (int x = 0; x < FACILITIES_LOCATIONS; x++) {
			sum += input_matrix[j + (i * FACILITIES_LOCATIONS)][x]
				* d_transposeDistancesMatrix[x][k];
		}
		output_matrix[j + (i * FACILITIES_LOCATIONS)][k] = sum;
	}
}

/**
 *  Multiplication between matrix a and matrix b
 */
__global__ void matrixMultiplication(unsigned int input_matrix_a[][FACILITIES_LOCATIONS],
	short input_matrix_b[][FACILITIES_LOCATIONS],
	unsigned int output_matrix[][FACILITIES_LOCATIONS]) {

	int i = blockIdx.x;
	int j = threadIdx.x;
	int k = threadIdx.y;

	if (j < FACILITIES_LOCATIONS && k < FACILITIES_LOCATIONS) {
		int sum = 0;
		#pragma unroll
		for (int x = 0; x < FACILITIES_LOCATIONS; x++) {
			sum += input_matrix_a[j + (i * FACILITIES_LOCATIONS)][x]
				* input_matrix_b[x + (i * FACILITIES_LOCATIONS)][k];
		}
		output_matrix[j + (i * FACILITIES_LOCATIONS)][k] = sum;
	}
}

__global__ void calculateTrace(short objective_id,
	unsigned int input_matrix[][FACILITIES_LOCATIONS],
	unsigned int d_population_fitness[][OBJECTIVES + 1],
	unsigned int d_sorted_population_fitness[][OBJECTIVES + 1]) {

	int i = blockIdx.x;
	unsigned int sum = 0;
	#pragma unroll
	for (int x = 0; x < FACILITIES_LOCATIONS; x++) {
		sum += input_matrix[x + (i * FACILITIES_LOCATIONS)][x];
	}
	d_population_fitness[i][objective_id] = d_sorted_population_fitness[i][objective_id] = sum;
}

__global__ void setOriginalIndex(unsigned int d_population_fitness[][OBJECTIVES + 1], unsigned int d_sorted_population_fitness[][OBJECTIVES + 1]) {

	d_population_fitness[blockIdx.x][OBJECTIVES] = d_sorted_population_fitness[blockIdx.x][OBJECTIVES] = blockIdx.x;
}

/**
 * This function calculates the fitness of each objective for all chromosomes in the 
 * population. The flow matrix is multiplied with the chromosome (represented in a 
 * binary 2d matrix), the resultant matrix is multiplied with the distance transposed
 * matrix, then the resultant matrix is multiplied with the transposed chromosome
 * (also a binary 2d matrix). The trace of this resultant matrix is the chromosome's
 * fitness. The fitness must be calculated for each flow matrix.
 * Trace(Fn*X*DT*XT)
 */
void parallelPopulationFitnessCalculation(
	short h_population[][FACILITIES_LOCATIONS], unsigned int h_population_fitness[][OBJECTIVES + 1],
	short d_population[][FACILITIES_LOCATIONS], unsigned int d_population_fitness[][OBJECTIVES + 1], unsigned int d_sorted_population_fitness[][OBJECTIVES + 1]) {

	/* Variable to check correct synchronization */
	hipError_t cudaStatus;

	/*******************************************************************************************
	 * Comment this section if you don't need to print partial results of fitness calculation.
	 */

	// Variable for population binary 2d representation in host memory (X).
	short h_2d_population[NSGA2_POPULATION_SIZE * FACILITIES_LOCATIONS][FACILITIES_LOCATIONS];

	// Variable for population binary 2d representation transposed in host memory (XT).
	short h_2d_transposed_population[NSGA2_POPULATION_SIZE * FACILITIES_LOCATIONS][FACILITIES_LOCATIONS];

	// Variable to keep Fn*X result in host memory (Fn: Flow matrix n).
	int h_temporal_1[NSGA2_POPULATION_SIZE * FACILITIES_LOCATIONS][FACILITIES_LOCATIONS];

	// Variable to keep Fn*X*DT result in host memory (DT: Transposed Distances matrix).
	int h_temporal_2[NSGA2_POPULATION_SIZE * FACILITIES_LOCATIONS][FACILITIES_LOCATIONS];

	// Variable to keep Fn*X*DT*XT result in host memory.
	int h_temporal_3[NSGA2_POPULATION_SIZE * FACILITIES_LOCATIONS][FACILITIES_LOCATIONS];

	/********************************************************************************************/

	// Variable for population binary 2d representation in device memory (X).
	short(*d_2d_population)[FACILITIES_LOCATIONS];
	hipMalloc((void**)&d_2d_population,
		sizeof(short) * NSGA2_POPULATION_SIZE * FACILITIES_LOCATIONS
		* FACILITIES_LOCATIONS);

	// Variable for population binary 2d representation transposed in device memory (XT).
	short(*d_2d_transposed_population)[FACILITIES_LOCATIONS];
	hipMalloc((void**)&d_2d_transposed_population,
		sizeof(short) * NSGA2_POPULATION_SIZE * FACILITIES_LOCATIONS
		* FACILITIES_LOCATIONS);

	/*
	 * Variable to keep Fn*X result in device memory (Fn: Flow matrix n).
	 * This variable is also used to keep Fn*X*DT*XT result.
	 */
	unsigned int(*d_temporal_1)[FACILITIES_LOCATIONS];
	hipMalloc((void**)&d_temporal_1,
		sizeof(unsigned int) * NSGA2_POPULATION_SIZE * FACILITIES_LOCATIONS
		* FACILITIES_LOCATIONS);

	// Variable to keep Fn*X*DT result in device memory (DT: Transposed Distances matrix).
	unsigned int(*d_temporal_2)[FACILITIES_LOCATIONS];
	hipMalloc((void**)&d_temporal_2,
		sizeof(unsigned int) * NSGA2_POPULATION_SIZE * FACILITIES_LOCATIONS
		* FACILITIES_LOCATIONS);
	
	for (int obj = 0; obj < OBJECTIVES; obj++) {

		dim3 threads(32, 32);
		populationTo2DRepresentation <<<NSGA2_POPULATION_SIZE, threads>>> (d_population,
			d_2d_population, d_2d_transposed_population);
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "populationTo2DRepresentation Sync CudaError!\n");
		}

		/*********************************************************************
		 * Comment this section if you don't need to print the partial results
		 * of fitness calculation.
		 */

		// Set current population binary 2d representation in host memory from device memory
		hipMemcpy(h_2d_population, d_2d_population,
		NSGA2_POPULATION_SIZE * FACILITIES_LOCATIONS * FACILITIES_LOCATIONS * sizeof(short),
				hipMemcpyDeviceToHost);
		// Set current population binary 2d representation transposed in host memory from device memory
		hipMemcpy(h_2d_transposed_population, d_2d_transposed_population,
		NSGA2_POPULATION_SIZE * FACILITIES_LOCATIONS * FACILITIES_LOCATIONS * sizeof(short),
				hipMemcpyDeviceToHost);
		/*********************************************************************/

		/*
		 * Fn*X
		 */
		multiplicationWithFlowMatrix <<<NSGA2_POPULATION_SIZE, threads>>> (obj,
			d_2d_population, d_temporal_1);
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "multiplicationWithFlowMatrix Sync CudaError!\n");
		}

		/*********************************************************************
		 * Comment this section if you don't need to print the partial results
		 * of fitness calculation.
		 */

		// Set the result of F1*X in host memory from device memory
		hipMemcpy(h_temporal_1, d_temporal_1,
			NSGA2_POPULATION_SIZE * FACILITIES_LOCATIONS * FACILITIES_LOCATIONS
						* sizeof(unsigned int), hipMemcpyDeviceToHost);
		/*********************************************************************/

		/*
		 * Fn*X*DT
		 */
		multiplicationWithTranposedDistanceMatrix <<<NSGA2_POPULATION_SIZE, threads>>> (
			d_temporal_1, d_temporal_2);
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "multiplicationWithTranposedDistanceMatrix Sync CudaError!\n");
		}

		/*********************************************************************
		 * Comment this section if you don't need to print the partial results
		 * of fitness calculation.
		 */

		// Set the result of Fn*X*DT in host memory from device memory
		hipMemcpy(h_temporal_2, d_temporal_2,
			NSGA2_POPULATION_SIZE * FACILITIES_LOCATIONS * FACILITIES_LOCATIONS * sizeof(unsigned int),
				hipMemcpyDeviceToHost);
		/*********************************************************************/
		
		/*
		 * Fn*X*DT*XT
		 */
		matrixMultiplication <<<NSGA2_POPULATION_SIZE, threads>>> (d_temporal_2,
			d_2d_transposed_population, d_temporal_1);
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "matrixMultiplication Sync CudaError!\n");
		}
		/*********************************************************************
		 * Comment this section if you don't need to print the partial results
		 * of fitness calculation.
		 */

		// Set the result of Fn*X*DT*XT in host memory from device memory
		hipMemcpy(h_temporal_3, d_temporal_1,
			NSGA2_POPULATION_SIZE * FACILITIES_LOCATIONS * FACILITIES_LOCATIONS * sizeof(unsigned int),
				hipMemcpyDeviceToHost);
		/*********************************************************************/

		/*
		 * Trace(Fn*X*DT*XT)
		 */
		calculateTrace <<<NSGA2_POPULATION_SIZE, 1 >>> (obj, d_temporal_1, d_population_fitness, d_sorted_population_fitness);

		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "calculateTrace 1 Sync CudaError!\n");
		}

		// Set original index. this is used in sorting operation.
		setOriginalIndex << <NSGA2_POPULATION_SIZE, 1 >> > (d_population_fitness, d_sorted_population_fitness);

		/* Set current population fitness in host memory from device memory */
		hipMemcpy(h_population_fitness, d_population_fitness,
			NSGA2_POPULATION_SIZE * (OBJECTIVES + 1) * sizeof(unsigned int),
			hipMemcpyDeviceToHost);

		/*********************************************************************
		 * Comment this section if you don't need to print the partial results
		 * of fitness calculation.
		 * NOTE: If you uncomment this section, all previous sections with this
		 * notes MUST be uncomented too.
		 */
		printf("\nPopulation IN FITNESS OPERATION\n");
		for (int i = 0; i < NSGA2_POPULATION_SIZE; i++) {
			printf("Chromosome %d\n", i);
			for (int j = 0; j < FACILITIES_LOCATIONS; j++) {
				printf("%d ", h_population[i][j]);
			}
			printf("\n");

			printf("\n2d Matrix Representation (X) \n");
			for (int x = 0; x < FACILITIES_LOCATIONS; x++) {

				for (int y = 0; y < FACILITIES_LOCATIONS; y++) {
					printf("%d ", h_2d_population[x + (i * FACILITIES_LOCATIONS)][y]);
				}
				printf("\n");
			}

			printf("\n2d Matrix Representation (Transposed) (XT)\n");
			for (int x = 0; x < FACILITIES_LOCATIONS; x++) {

				for (int y = 0; y < FACILITIES_LOCATIONS; y++) {
					printf("%d ", h_2d_transposed_population[x + (i * FACILITIES_LOCATIONS)][y]);
				}
				printf("\n");
			}

			printf("\n F%d*X \n", obj);
			for (int x = 0; x < FACILITIES_LOCATIONS; x++) {

				for (int y = 0; y < FACILITIES_LOCATIONS; y++) {
					printf("%d ", h_temporal_1[x + (i * FACILITIES_LOCATIONS)][y]);
				}
				printf("\n");
			}

			printf("\n F%d*X*DT \n", obj);
			for (int x = 0; x < FACILITIES_LOCATIONS; x++) {
				for (int y = 0; y < FACILITIES_LOCATIONS; y++) {
					printf("%d ", h_temporal_2[x + (i * FACILITIES_LOCATIONS)][y]);
				}
				printf("\n");
			}

			printf("\n F%d*X*DT*XT \n", obj);
			for (int x = 0; x < FACILITIES_LOCATIONS; x++) {
				for (int y = 0; y < FACILITIES_LOCATIONS; y++) {
					printf("%d ", h_temporal_3[x + (i * FACILITIES_LOCATIONS)][y]);
				}
				printf("\n");
			}
			printf("\n");
		}
		/*********************************************************************/
	}

	hipFree(d_2d_population);
	hipFree(d_2d_transposed_population);
	hipFree(d_temporal_1);
	hipFree(d_temporal_2);
	
}


/**
 * Set initial values to total dominance, rank and crowding variables (fill with zeros).
 */
__global__ void initializeNSGA2Variables(
	short d_population_total_dominance[], short d_population_rank[], float d_population_crowding[], 
	unsigned int d_population_fitness[][OBJECTIVES + 1], float d_temporal_population_crowding[][3]) {
	d_population_total_dominance[blockIdx.x] = 0;
	d_population_rank[blockIdx.x] = 0;
	d_population_crowding[blockIdx.x] = 0;
	d_temporal_population_crowding[blockIdx.x][0] = 0;
	d_temporal_population_crowding[blockIdx.x][1] = 0;
	d_temporal_population_crowding[blockIdx.x][2] = 0;

	/*
	 * Test values to verify the Pareto fronts calculation, works with POPULATION = 9
	 * must be removed after verification, also d_population_fitness must be removed
	 * in kernel calls and definition
	 *
	d_population_fitness[0][0] = 10;
	d_population_fitness[0][1] = 625;
	d_population_fitness[1][0] = 40;
	d_population_fitness[1][1] = 600;
	d_population_fitness[2][0] = 30;
	d_population_fitness[2][1] = 500;
	d_population_fitness[3][0] = 0;
	d_population_fitness[3][1] = 400;
	d_population_fitness[4][0] = 20;
	d_population_fitness[4][1] = 325;
	d_population_fitness[5][0] = 60;
	d_population_fitness[5][1] = 450;
	d_population_fitness[6][0] = 70;
	d_population_fitness[6][1] = 375;
	d_population_fitness[7][0] = 60;
	d_population_fitness[7][1] = 275;
	d_population_fitness[8][0] = 80;
	d_population_fitness[8][1] = 125;


	d_population_fitness[9][0] = 100;
	d_population_fitness[9][1] = 0;
	d_population_fitness[10][0] = 90;
	d_population_fitness[10][1] = 290;
	d_population_fitness[11][0] = 100;
	d_population_fitness[11][1] = 400;
	d_population_fitness[12][0] = 120;
	d_population_fitness[12][1] = 375;
	d_population_fitness[13][0] = 140;
	d_population_fitness[13][1] = 350;
	d_population_fitness[14][0] = 150;
	d_population_fitness[14][1] = 250;
	d_population_fitness[15][0] = 170;
	d_population_fitness[15][1] = 75;
	d_population_fitness[16][0] = 170;
	d_population_fitness[16][1] = 300;
	d_population_fitness[17][0] = 180;
	d_population_fitness[17][1] = 50;
	/*****************************************/
}


/**
 * This function get the population dominancen matrix for 2-objective problems.
 */
__global__ void get2ObjectivePopulationDominanceMatrix(bool d_population_dominance_matrix[][NSGA2_POPULATION_SIZE], unsigned int d_population_fitness[][OBJECTIVES + 1]) {

	d_population_dominance_matrix[blockIdx.x][threadIdx.x] =
	(
		(d_population_fitness[threadIdx.x][0] <= d_population_fitness[blockIdx.x][0]) &&
		(d_population_fitness[threadIdx.x][1] <= d_population_fitness[blockIdx.x][1])
	) &&
	(
		(d_population_fitness[threadIdx.x][0] < d_population_fitness[blockIdx.x][0]) ||
		(d_population_fitness[threadIdx.x][1] < d_population_fitness[blockIdx.x][1])
	);
}

/**
 * This function get the population dominancen matrix for 3-objective problems.
 */
__global__ void get3ObjectivePopulationDominanceMatrix(bool d_population_dominance_matrix[][NSGA2_POPULATION_SIZE], unsigned int d_population_fitness[][OBJECTIVES + 1]) {

	d_population_dominance_matrix[blockIdx.x][threadIdx.x] =
	(
		(d_population_fitness[threadIdx.x][0] <= d_population_fitness[blockIdx.x][0]) &&
		(d_population_fitness[threadIdx.x][1] <= d_population_fitness[blockIdx.x][1]) &&
		(d_population_fitness[threadIdx.x][2] <= d_population_fitness[blockIdx.x][2])
	) &&
	(
		(d_population_fitness[threadIdx.x][0] < d_population_fitness[blockIdx.x][0]) ||
		(d_population_fitness[threadIdx.x][1] < d_population_fitness[blockIdx.x][1]) ||
		(d_population_fitness[threadIdx.x][2] < d_population_fitness[blockIdx.x][2])
	);

}

__global__ void getParallelTotalDominance(short d_population_total_dominance[], bool d_population_dominance_matrix[][NSGA2_POPULATION_SIZE]) {

	unsigned int sum = 0;
	#pragma unroll
	for (int x = 0; x < NSGA2_POPULATION_SIZE; x++) {
		sum += d_population_dominance_matrix[blockIdx.x][x];
	}
	d_population_total_dominance[blockIdx.x] = sum;
}

__global__ void setRank(int iteration, short d_population_total_dominance[], short d_population_rank[]) {

	if (d_population_total_dominance[blockIdx.x] == 0 && d_population_rank[blockIdx.x] == 0) {
		d_population_rank[blockIdx.x] = iteration;
	}

}

__global__ void cleanDominanceMatrix(int iteration, bool d_population_dominance_matrix[][NSGA2_POPULATION_SIZE], short d_population_rank[]) {

	if (d_population_rank[threadIdx.x] == iteration) {
		d_population_dominance_matrix[blockIdx.x][threadIdx.x] = 0;
	}

}

__global__ void bitonicSortStep(unsigned int d_population_fitness[][OBJECTIVES + 1], int j, int k, short objective) {
	unsigned int i, ixj;
	i = threadIdx.x + blockDim.x * blockIdx.x;
	ixj = i ^ j;
	// Threads with the lowest ids sort the array.
	if ((ixj) > i) {
		if ((i & k) == 0) {
			// Sorting ascending.
			if (d_population_fitness[i][objective] > d_population_fitness[ixj][objective]) {
				unsigned int temp0 = d_population_fitness[i][0];
				unsigned int temp1 = d_population_fitness[i][1];
				unsigned int temp2 = d_population_fitness[i][2];

				d_population_fitness[i][0] = d_population_fitness[ixj][0];
				d_population_fitness[i][1] = d_population_fitness[ixj][1];
				d_population_fitness[i][2] = d_population_fitness[ixj][2];

				d_population_fitness[ixj][0] = temp0;
				d_population_fitness[ixj][1] = temp1;
				d_population_fitness[ixj][2] = temp2;

				if (OBJECTIVES == 3) {
					unsigned int temp3= d_population_fitness[i][3];
					d_population_fitness[i][3] = d_population_fitness[ixj][3];
					d_population_fitness[ixj][3] = temp3;
				}
			}
		}
		if ((i & k) != 0) {
			// Sorting descending.
			if (d_population_fitness[i][objective] < d_population_fitness[ixj][objective]) {

				unsigned int temp0 = d_population_fitness[i][0];
				unsigned int temp1 = d_population_fitness[i][1];
				unsigned int temp2 = d_population_fitness[i][2];

				d_population_fitness[i][0] = d_population_fitness[ixj][0];
				d_population_fitness[i][1] = d_population_fitness[ixj][1];
				d_population_fitness[i][2] = d_population_fitness[ixj][2];

				d_population_fitness[ixj][0] = temp0;
				d_population_fitness[ixj][1] = temp1;
				d_population_fitness[ixj][2] = temp2;

				if (OBJECTIVES == 3) {
					unsigned int temp3 = d_population_fitness[i][3];
					d_population_fitness[i][3] = d_population_fitness[ixj][3];
					d_population_fitness[ixj][3] = temp3;
				}
			}
		}
	}
}

/**
 * Desc population sort by the input objective using bitonic sort.
 */
void sortPopulationByFitness(unsigned int d_population_fitness[][OBJECTIVES + 1], unsigned int objective)
{
	// Variable for population fitness in host memory.
	unsigned int h_population_fitness[NSGA2_POPULATION_SIZE][OBJECTIVES + 1];

	/* Set current population fitness in host memory from device memory */
	hipMemcpy(h_population_fitness, d_population_fitness,
		NSGA2_POPULATION_SIZE * (OBJECTIVES + 1) * sizeof(unsigned int),
		hipMemcpyDeviceToHost);

	printf("\nUNSORTED fitness ojective %d \n", objective);
	for (int i = 0; i < NSGA2_POPULATION_SIZE; i++) {
		for (int j = 0; j < OBJECTIVES + 1; j++) {
			printf("|%d ", h_population_fitness[i][j]);
		}
		printf("\n");
	}

	int j, k;
	// Major step.
	for (k = 2; k <= NSGA2_POPULATION_SIZE; k <<= 1) {
		// Minor step.
		for (j = k >> 1; j > 0; j = j >> 1) {
			bitonicSortStep <<<POPULATION_SIZE, 2 >> > (d_population_fitness, j, k, objective);
		}
	}

	/* Set current population fitness in host memory from device memory */
	hipMemcpy(h_population_fitness, d_population_fitness,
		NSGA2_POPULATION_SIZE * (OBJECTIVES + 1) * sizeof(unsigned int),
		hipMemcpyDeviceToHost);

	printf("\nSORTED fitness ojective %d \n", objective);
	for (int i = 0; i < NSGA2_POPULATION_SIZE; i++) {
		for (int j = 0; j < OBJECTIVES + 1; j++) {
			printf("|%d ", h_population_fitness[i][j]);
		}
		printf("\n");
	}

}

__global__ void crowdingCalcualtion(
	short current_pareto_front, float d_temporal_population_crowding[][3], float d_population_crowding[],
	unsigned int d_population_fitness[][OBJECTIVES + 1], unsigned int d_sorted_population_fitness[][OBJECTIVES + 1],
	short objective) {

	
	if (blockIdx.x == 0 || 
		((blockIdx.x + 1) < NSGA2_POPULATION_SIZE && (blockIdx.x - 1) >= 0 &&
		 d_temporal_population_crowding[blockIdx.x + 1][0] != current_pareto_front &&
		 d_temporal_population_crowding[blockIdx.x][0] == current_pareto_front)
		) {
		 
		d_temporal_population_crowding[blockIdx.x][2] = (unsigned int)HUGE_VALF;
	}
	else if ((blockIdx.x + 1) < NSGA2_POPULATION_SIZE && d_temporal_population_crowding[blockIdx.x + 1][0] == current_pareto_front) {
		d_temporal_population_crowding[blockIdx.x][2] = 
			(float)d_temporal_population_crowding[blockIdx.x][2] + 
			(
				(float)(d_population_fitness[(int)d_temporal_population_crowding[blockIdx.x + 1][1]][objective] - 
				 d_population_fitness[(int)d_temporal_population_crowding[blockIdx.x - 1][1]][objective]) /
				(float)(d_sorted_population_fitness[NSGA2_POPULATION_SIZE - 1][objective] - d_sorted_population_fitness[0][objective])
			);
	}
	d_population_crowding[(int)d_temporal_population_crowding[blockIdx.x][1]] = d_temporal_population_crowding[blockIdx.x][2];

}

/*
 * Set as zero all initial crowding distances.
 */
__global__ void crowdingInitialization(float d_temporal_population_crowding[][3], float d_population_crowding[]) {
	d_population_crowding[blockIdx.x] = 0;
	d_temporal_population_crowding[blockIdx.x][2] = 0;
}


__global__ void paretoFrontBitonicSortStep(float d_temporal_population_crowding[][3], int j, int k) {
	unsigned int i, ixj;
	i = threadIdx.x + blockDim.x * blockIdx.x;
	ixj = i ^ j;
	// Threads with the lowest ids sort the array.
	if ((ixj) > i) {
		if ((i & k) == 0) {
			// Sorting ascending.
			if (d_temporal_population_crowding[i][2] > d_temporal_population_crowding[ixj][2]) {
				float temp0 = d_temporal_population_crowding[i][0];
				float temp1 = d_temporal_population_crowding[i][1];
				float temp2 = d_temporal_population_crowding[i][2];

				d_temporal_population_crowding[i][0] = d_temporal_population_crowding[ixj][0];
				d_temporal_population_crowding[i][1] = d_temporal_population_crowding[ixj][1];
				d_temporal_population_crowding[i][2] = d_temporal_population_crowding[ixj][2];

				d_temporal_population_crowding[ixj][0] = temp0;
				d_temporal_population_crowding[ixj][1] = temp1;
				d_temporal_population_crowding[ixj][2] = temp2;

			}
		}
		if ((i & k) != 0) {
			// Sorting descending.
			if (d_temporal_population_crowding[i][2] < d_temporal_population_crowding[ixj][2]) {

				float temp0 = d_temporal_population_crowding[i][0];
				float temp1 = d_temporal_population_crowding[i][1];
				float temp2 = d_temporal_population_crowding[i][2];

				d_temporal_population_crowding[i][0] = d_temporal_population_crowding[ixj][0];
				d_temporal_population_crowding[i][1] = d_temporal_population_crowding[ixj][1];
				d_temporal_population_crowding[i][2] = d_temporal_population_crowding[ixj][2];

				d_temporal_population_crowding[ixj][0] = temp0;
				d_temporal_population_crowding[ixj][1] = temp1;
				d_temporal_population_crowding[ixj][2] = temp2;
			}
		}
	}
}


/**
 * Desc Pareto front sort by the crowding distance using bitonic sort.
 */
void sortParetoFrontByCrowding(float d_temporal_population_crowding[][3])
{
	int j, k;
	// Major step.
	for (k = 2; k <= NSGA2_POPULATION_SIZE; k <<= 1) {
		// Minor step.
		for (j = k >> 1; j > 0; j = j >> 1) {
			paretoFrontBitonicSortStep << <POPULATION_SIZE, 2 >> > (d_temporal_population_crowding, j, k);
		}
	}
}

/*
 * Caculates the crowding distance of all elements in a Parteto Front
 */

short crowding(
	short current_pareto_front,
	unsigned int d_population_fitness[][OBJECTIVES + 1], 
	unsigned int d_sorted_population_fitness[][OBJECTIVES + 1],
	short d_population_rank[],
	short h_population_rank[],
	float d_temporal_population_crowding[][3],
	float d_population_crowding[],
	float h_population_crowding[]) {

	// Variable to store temporarily the Pareto front crowding in host memory.
	// [0] for rank.
	// [1] original index.
	// [2] crowding distance.
	float h_temporal_population_crowding[NSGA2_POPULATION_SIZE][3];

	// Variable for sorted population fitness in host memory.
	unsigned int h_sorted_population_fitness[NSGA2_POPULATION_SIZE][OBJECTIVES + 1] ;

	short top_total;
	short botton_total;

	crowdingInitialization <<<NSGA2_POPULATION_SIZE, 1 >>> (d_temporal_population_crowding, d_population_crowding);

	/* Set current Pareto front crowding in host memory from device memory */
	hipMemcpy(h_temporal_population_crowding, d_temporal_population_crowding,
		NSGA2_POPULATION_SIZE * 3 * sizeof(float),
		hipMemcpyDeviceToHost);
	
	hipMemcpy(h_population_crowding, d_population_crowding,
		NSGA2_POPULATION_SIZE * sizeof(float),
		hipMemcpyDeviceToHost);
	
	for (short objective = 0; objective < OBJECTIVES; objective++) {
		sortPopulationByFitness(d_sorted_population_fitness, objective);
		/* Set current Sorted by F(i-objective) population in host memory from device memory */
		hipMemcpy(h_sorted_population_fitness, d_sorted_population_fitness,
			NSGA2_POPULATION_SIZE * (OBJECTIVES + 1) * sizeof(unsigned int),
			hipMemcpyDeviceToHost);

		top_total = botton_total = 0;

		for (short i = 0; i < NSGA2_POPULATION_SIZE; i++) {

			if (h_population_rank[h_sorted_population_fitness[i][OBJECTIVES]] == current_pareto_front) {
				h_temporal_population_crowding[top_total][0] = current_pareto_front;
				h_temporal_population_crowding[top_total][1] = h_sorted_population_fitness[i][OBJECTIVES];
				h_temporal_population_crowding[top_total][2] = h_population_crowding[h_sorted_population_fitness[i][OBJECTIVES]];
				top_total++;
			}
			else {
				h_temporal_population_crowding[NSGA2_POPULATION_SIZE - (botton_total + 1)][0] = h_population_rank[h_sorted_population_fitness[i][OBJECTIVES]];
				h_temporal_population_crowding[NSGA2_POPULATION_SIZE - (botton_total + 1)][1] = h_sorted_population_fitness[i][OBJECTIVES];
				h_temporal_population_crowding[NSGA2_POPULATION_SIZE - (botton_total + 1)][2] = h_population_crowding[h_sorted_population_fitness[i][OBJECTIVES]];
				botton_total++;
			}
		}

		printf("\nTEMPORAL(INICIO) Pareto front %d OBJECTIVE %d \n", current_pareto_front, objective);
		for (int x = 0; x < NSGA2_POPULATION_SIZE; x++) {
			for (int y = 0; y < 3; y++) {
				printf("%f ", h_temporal_population_crowding[x][y]);
			}
			printf("\n");
		}

		// Set population crowding in device memory from host memory.
		hipMemcpy(d_temporal_population_crowding, h_temporal_population_crowding,
			NSGA2_POPULATION_SIZE * 3 * sizeof(float),
			hipMemcpyHostToDevice);

		crowdingCalcualtion << <NSGA2_POPULATION_SIZE, 1 >> > (current_pareto_front, d_temporal_population_crowding, d_population_crowding, d_population_fitness, d_sorted_population_fitness, objective);
		
		/* Set population crowding in host memory from device memory */
		hipMemcpy(h_temporal_population_crowding, d_temporal_population_crowding,
			NSGA2_POPULATION_SIZE * 3 * sizeof(float),
			hipMemcpyDeviceToHost);
		hipMemcpy(h_population_crowding, d_population_crowding,
			NSGA2_POPULATION_SIZE * sizeof(float),
			hipMemcpyDeviceToHost);

		printf("\nTEMPORAL(FIN) Pareto front %d OBJECTIVE %d \n", current_pareto_front, objective);
		for (int x = 0; x < NSGA2_POPULATION_SIZE; x++) {
			for (int y = 0; y < 3; y++) {
				printf("%f ", h_temporal_population_crowding[x][y]);
			}
			printf("\n");
		}

	}

	sortParetoFrontByCrowding(d_temporal_population_crowding);

	return top_total;
}

void parallelNSGA2(
	short h_population[][FACILITIES_LOCATIONS], unsigned int h_population_fitness[][OBJECTIVES + 1],
	short h_population_total_dominance[], short h_population_rank[], float h_population_crowding[],
	short d_population[][FACILITIES_LOCATIONS], unsigned int d_population_fitness[][OBJECTIVES + 1], unsigned int d_sorted_population_fitness[][OBJECTIVES + 1],
	short d_population_total_dominance[], short d_population_rank[], float d_population_crowding[]) {

	/* Variable to check correct synchronization */
	hipError_t cudaStatus;

	/*******************************************************************************************
	 * Comment this section if you don't need to print partial results of NSGA2.
	 */

	// Variable to store the population dominance in host memory.
	bool h_population_dominance_matrix[NSGA2_POPULATION_SIZE][NSGA2_POPULATION_SIZE];

	/*******************************************************************************************/

	// Variable to store temporarily the Pareto front crowding in host memory.
	// [0] for rank.
	// [1] original index.
	// [2] crowding distance.
	float h_temporal_population_crowding[NSGA2_POPULATION_SIZE][3];

	// Variable to store the the total count of selected solution in each iteration.
	short offspring_count = 0;

	// Number of solutions in the Parto front.
	short pareto_front_count = 0;

	// Variable to store the population dominancen in device memory.
	bool(*d_population_dominance_matrix)[NSGA2_POPULATION_SIZE];
	hipMalloc((void**)&d_population_dominance_matrix,
		sizeof(bool) * NSGA2_POPULATION_SIZE * NSGA2_POPULATION_SIZE);

	// Variable to store temporarily the population crowding in device memory.
	// will be use for crauding calculation and offspring selection
	// [0] for rank.
	// [1] original index.
	// [2] crowding distance.
	float(*d_temporal_population_crowding)[3];
	hipMalloc((void**)&d_temporal_population_crowding,
		sizeof(float) * NSGA2_POPULATION_SIZE * 3);

	/*
	 * Set initial values to totaldominance, rank and crowding variables (fill with zeros).
	 */
	initializeNSGA2Variables <<<NSGA2_POPULATION_SIZE, 1 >>> (d_population_total_dominance, d_population_rank, d_population_crowding, d_population_fitness, d_temporal_population_crowding);

	/*
	 * calculate the populaton dominace matrix.
	 */
	if (OBJECTIVES == 2) {
		get2ObjectivePopulationDominanceMatrix <<<NSGA2_POPULATION_SIZE, NSGA2_POPULATION_SIZE>>> (d_population_dominance_matrix, d_population_fitness);
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "get2ObjectivePopulationDominanceMatrix Sync CudaError!\n");
		}
	}
	else if (OBJECTIVES == 3) {
		get3ObjectivePopulationDominanceMatrix <<<NSGA2_POPULATION_SIZE, NSGA2_POPULATION_SIZE>>> (d_population_dominance_matrix, d_population_fitness);
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "get3ObjectivePopulationDominanceMatrix Sync CudaError!\n");
		}
	}
	else {
		printf("\nThis solution only suports 2 and 3 objetives QAP\n");
		exit(0);
	}

	/*
	 * calculate the total dominance per solution.
	 */
	getParallelTotalDominance <<<NSGA2_POPULATION_SIZE, 1 >>> (d_population_total_dominance, d_population_dominance_matrix);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "getParallelTotalDominance Sync CudaError!\n");
	}


	/*********************************************************************
	 * Comment this section if you don't need to print the partial results
	 * of  population dominace matrix calculation.
	 */

	// Set current population dominance matrix in host memory from device memory.
	hipMemcpy(h_population_dominance_matrix, d_population_dominance_matrix,
		NSGA2_POPULATION_SIZE * NSGA2_POPULATION_SIZE * sizeof(bool),
		hipMemcpyDeviceToHost);
	// Set current population total dominance in host memory from device memory.
	hipMemcpy(h_population_total_dominance, d_population_total_dominance,
		NSGA2_POPULATION_SIZE * sizeof(short),
		hipMemcpyDeviceToHost);

	printf("\nPopulation dominace matrix\n");
	for (int i = 0; i < NSGA2_POPULATION_SIZE; i++) {
		for (int j = 0; j < NSGA2_POPULATION_SIZE; j++) {
			printf("%d ", h_population_dominance_matrix[i][j]);
		}
		printf("| %d\n", h_population_total_dominance[i]);
		printf("\n");
	}
	/*********************************************************************/


	// Main Routine to calculate Pareto Fronts (until NSGA2_POPULATION_SIZE, the worst case or offspring population completed).
	for (int i = 1; i < NSGA2_POPULATION_SIZE; i++) {
		// Set NSGA2 Rank.
		setRank <<<NSGA2_POPULATION_SIZE,1 >>> (i, d_population_total_dominance, d_population_rank);
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "setRank Sync CudaError!\n");
		}

		// Set current population ranck in host memory from device memory. (MANDATORY) 
		hipMemcpy(h_population_rank, d_population_rank,
			NSGA2_POPULATION_SIZE * sizeof(short),
			hipMemcpyDeviceToHost);

		/*********************************************************************/
		printf("\nPopulation Rank \n");
		for (int x = 0; x < NSGA2_POPULATION_SIZE; x++) {
			printf("%d \n", h_population_rank[x]);
		}
		/*********************************************************************/

		// Calculate Pareto front Crowding distance
		pareto_front_count = crowding(i, d_population_fitness, d_sorted_population_fitness, d_population_rank, h_population_rank, d_temporal_population_crowding, d_population_crowding, h_population_crowding);

		/*********************************************************************
		 * Comment this section if you don't need to print the partial results
		 * of Pareto front sorted by crowding distance.
		 */

		 // Set current Pareto front sorted by crowding distance in host memory from device memory.
		hipMemcpy(h_temporal_population_crowding, d_temporal_population_crowding,
			NSGA2_POPULATION_SIZE * 3 * sizeof(float),
			hipMemcpyDeviceToHost);

		printf("\nPareto front %d sorted by crowding distance\n", i);
		for (int x = 0; x < NSGA2_POPULATION_SIZE; x++) {
			for (int y = 0; y < 3; y++) {
				printf("%f ", h_temporal_population_crowding[x][y]);
			}
			printf("\n");
		}
		/*********************************************************************/

		// @todo Move selected pareto front solutions to the offspring.
		//moveSolutionsToOffspring <<<NSGA2_POPULATION_SIZE, pareto_front_count >> > (offspring_count, pareto_front_count, d_population, d_temporal_population_crowding);
		//moveRankToOffspring <<<NSGA2_POPULATION_SIZE, 1 >> > (offspring_count, pareto_front_count, d_rank, d_temporal_population_crowding);
		//moveCrowdingToOffspring <<<NSGA2_POPULATION_SIZE, 1 >> > (offspring_count, pareto_front_count, d_population_crowding, d_temporal_population_crowding);

		offspring_count += pareto_front_count;

		printf("\nPareto front %d count: %d\n", i, pareto_front_count);
		printf("\nPareto front %d OFFSpring count: %d\n", i, offspring_count);

		// End the loop if the offspring is complete.
		if (offspring_count >= POPULATION_SIZE) {
			break;
		}

		// Remove the current Pareto Front elements from the dominance matrix.
		cleanDominanceMatrix <<<NSGA2_POPULATION_SIZE, NSGA2_POPULATION_SIZE >>> (i, d_population_dominance_matrix, d_population_rank);
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "cleanDominanceMatrix Sync CudaError!\n");
		}

		// Calculate the new total dominance per solution.
		getParallelTotalDominance <<<NSGA2_POPULATION_SIZE, 1 >>> (d_population_total_dominance, d_population_dominance_matrix);
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "getParallelTotalDominance in NSGA2 iteration Sync CudaError!\n");
		}

	}
}



int main()
{
	// In NSGA2, the Rt population size.
	//const int NSGA2_POPULATION_SIZE = POPULATION_SIZE * 2;
	// To measure the execution time.
	clock_t begin = clock();
	// To set seed variable.
	time_t t;
	// To check correct synchronization.
	hipError_t cudaStatus;
	// Initializes random number generator.
	srand((unsigned)time(&t));
	// Seed for hiprand.
	int seed = rand() % 10000;

	/***********VARIABLES IN HOST MEMORY*********/
	// Variable for population  in host memory.
	short h_population[NSGA2_POPULATION_SIZE][FACILITIES_LOCATIONS];
	// Variable for population fitness in host memory.
	unsigned int h_population_fitness[NSGA2_POPULATION_SIZE][OBJECTIVES + 1];
	// Variable for population total dominace in host memory.
	short h_population_total_dominance[NSGA2_POPULATION_SIZE];
	// Variable for population rank in host memory.
	short h_population_rank[NSGA2_POPULATION_SIZE];
	// Variable for population crowding distance in host memory.
	float h_population_crowding[NSGA2_POPULATION_SIZE];

	/**********VARIABLES IN DEVICE MEMORY********/
	// Variable for population in device memory.
	short(*d_population)[FACILITIES_LOCATIONS];
	hipMalloc((void**)&d_population, sizeof(short) * NSGA2_POPULATION_SIZE * FACILITIES_LOCATIONS);
	// Variable for population fitness in device memory.
	unsigned int(*d_population_fitness)[OBJECTIVES + 1];
	hipMalloc((void**)&d_population_fitness, sizeof(unsigned int) * NSGA2_POPULATION_SIZE * (OBJECTIVES + 1));
	// Variable for sorted population fitness in device memory.
	unsigned int(*d_sorted_population_fitness)[OBJECTIVES + 1];
	hipMalloc((void**)&d_sorted_population_fitness, sizeof(unsigned int) * NSGA2_POPULATION_SIZE * (OBJECTIVES + 1));
	// Variable for population total dominace in device memory.
	short(*d_population_total_dominance);
	hipMalloc((void**)&d_population_total_dominance, sizeof(short) * NSGA2_POPULATION_SIZE);
	// Variable for population rank in device memory.
	short(*d_population_rank);
	hipMalloc((void**)&d_population_rank, sizeof(short) * NSGA2_POPULATION_SIZE);
	// Variable for population crowding distance in device memory.
	float(*d_population_crowding);
	hipMalloc((void**)&d_population_crowding, sizeof(float) * NSGA2_POPULATION_SIZE);

	/* Generation of all base chromosomes (genes ordered ascending).
	 * 64 threads are defined here because we are going to tackle instances upto 60 FACILITIES/LOCATIONS.
	 */
	generateBasePopulation <<<NSGA2_POPULATION_SIZE, 64 >>> (d_population);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "generateBasePopulation Sync CudaError!\n");
	}

	/* Uncomment this section of code to print the base population
	 Set population in host memory from device memory */
	hipMemcpy(h_population, d_population,
		NSGA2_POPULATION_SIZE * (FACILITIES_LOCATIONS) * sizeof(short),
		hipMemcpyDeviceToHost);
	printf("\nBase Population\n");
	for (int i = 0; i < NSGA2_POPULATION_SIZE; i++) {
		printf("Chromosome %d\n", i);
		for (int j = 0; j < FACILITIES_LOCATIONS; j++) {
			printf("%d ", h_population[i][j]);
		}
		printf("\n");
	}
	/* */

	 /* Initialize variables for random values generation with hiprand */
	hiprandState* d_state;
	hipMalloc(&d_state, sizeof(hiprandState));
	unsigned* d_max_rand_int, * h_max_rand_int, * d_min_rand_int, * h_min_rand_int;
	hipMalloc(&d_max_rand_int, sizeof(unsigned));
	h_max_rand_int = (unsigned*)malloc(sizeof(unsigned));
	hipMalloc(&d_min_rand_int, sizeof(unsigned));
	h_min_rand_int = (unsigned*)malloc(sizeof(unsigned));
	*h_max_rand_int = FACILITIES_LOCATIONS;
	*h_min_rand_int = 0;
	hipMemcpy(d_max_rand_int, h_max_rand_int, sizeof(unsigned),
		hipMemcpyHostToDevice);
	hipMemcpy(d_min_rand_int, h_min_rand_int, sizeof(unsigned),
		hipMemcpyHostToDevice);

	// 64 threads are defined here because we are going to tackle instances upto 60 FACILITIES/LOCATIONS.
	curand_setup <<<NSGA2_POPULATION_SIZE, 64 >>> (d_state, seed);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "curand_setup Sync CudaError!");
	}

	/* Shuffles chromosome genes randomly over all population */
	shufflePopulationGenes <<<NSGA2_POPULATION_SIZE, 1 >>> (d_state, d_max_rand_int,
		d_min_rand_int, d_population);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "shufflePopulationGenes Sync CudaError!");
	}
	/* Uncomment this section of code to print the Shuffled population
	 Set current population (with Shuffled genes) in host memory from device memory */
	hipMemcpy(h_population, d_population,
		NSGA2_POPULATION_SIZE * FACILITIES_LOCATIONS * sizeof(short),
		hipMemcpyDeviceToHost);

	printf("\nShuffled Population\n");
	for (int i = 0; i < NSGA2_POPULATION_SIZE; i++) {
		printf("Chromosome %d\n", i);
		for (int j = 0; j < FACILITIES_LOCATIONS; j++) {
			printf("%d ", h_population[i][j]);
		}
		printf("\n");
	}
	/**/


	/* Set all chromosomes with 1 2 7 9 6 5 0 4 3 8 0 0 for test purposes 
	 * expected fitness: F0 = 228322 F1 = 193446
	 *
	
	for (int a = 0; a < POPULATION_SIZE; a++) {
		h_population[a][0] = 1;
		h_population[a][1] = 2;
		h_population[a][2] = 7;
		h_population[a][3] = 9;
		h_population[a][4] = 6;
		h_population[a][5] = 5;
		h_population[a][6] = 0;
		h_population[a][7] = 4;
		h_population[a][8] = 3;
		h_population[a][9] = 8;	
	}

	*/

	h_population[0][0] =1; h_population[0][1] =9; h_population[0][2] =6; h_population[0][3] =2; h_population[0][4] =8; h_population[0][5] =0; h_population[0][6] =4; h_population[0][7] =3; h_population[0][8] =5; h_population[0][9] =7;
	h_population[1][0] =3; h_population[1][1] =7; h_population[1][2] =4; h_population[1][3] =6; h_population[1][4] =2; h_population[1][5] =9; h_population[1][6] =1; h_population[1][7] =8; h_population[1][8] =0; h_population[1][9] =5;
	h_population[2][0] =4; h_population[2][1] =9; h_population[2][2] =8; h_population[2][3] =5; h_population[2][4] =6; h_population[2][5] =0; h_population[2][6] =7; h_population[2][7] =3; h_population[2][8] =2; h_population[2][9] =1;
	h_population[3][0] =2; h_population[3][1] =9; h_population[3][2] =1; h_population[3][3] =5; h_population[3][4] =4; h_population[3][5] =0; h_population[3][6] =3; h_population[3][7] =8; h_population[3][8] =6; h_population[3][9] =7;
	h_population[4][0] =6; h_population[4][1] =9; h_population[4][2] =0; h_population[4][3] =1; h_population[4][4] =8; h_population[4][5] =4; h_population[4][6] =3; h_population[4][7] =2; h_population[4][8] =5; h_population[4][9] =7;
	h_population[5][0] =8; h_population[5][1] =7; h_population[5][2] =5; h_population[5][3] =1; h_population[5][4] =4; h_population[5][5] =6; h_population[5][6] =9; h_population[5][7] =2; h_population[5][8] =0; h_population[5][9] =3;
	h_population[6][0] =4; h_population[6][1] =9; h_population[6][2] =3; h_population[6][3] =6; h_population[6][4] =0; h_population[6][5] =7; h_population[6][6] =8; h_population[6][7] =5; h_population[6][8] =2; h_population[6][9] =1;
	h_population[7][0] =3; h_population[7][1] =5; h_population[7][2] =1; h_population[7][3] =6; h_population[7][4] =7; h_population[7][5] =0; h_population[7][6] =9; h_population[7][7] =2; h_population[7][8] =8; h_population[7][9] =4;
	h_population[8][0] =6; h_population[8][1] =9; h_population[8][2] =1; h_population[8][3] =8; h_population[8][4] =2; h_population[8][5] =5; h_population[8][6] =7; h_population[8][7] =0; h_population[8][8] =4; h_population[8][9] =3;
	h_population[9][0] =4; h_population[9][1] =8; h_population[9][2] =7; h_population[9][3] =6; h_population[9][4] =9; h_population[9][5] =5; h_population[9][6] =1; h_population[9][7] =0; h_population[9][8] =3; h_population[9][9] =2;
	h_population[10][0] =8; h_population[10][1] =5; h_population[10][2] =4; h_population[10][3] =2; h_population[10][4] =0; h_population[10][5] =1; h_population[10][6] =7; h_population[10][7] =9; h_population[10][8] =3; h_population[10][9] =6;
	h_population[11][0] =3; h_population[11][1] =8; h_population[11][2] =0; h_population[11][3] =6; h_population[11][4] =2; h_population[11][5] =7; h_population[11][6] =4; h_population[11][7] =9; h_population[11][8] =5; h_population[11][9] =1;
	h_population[12][0] =1; h_population[12][1] =6; h_population[12][2] =5; h_population[12][3] =9; h_population[12][4] =8; h_population[12][5] =3; h_population[12][6] =0; h_population[12][7] =2; h_population[12][8] =4; h_population[12][9] =7;
	h_population[13][0] =2; h_population[13][1] =6; h_population[13][2] =3; h_population[13][3] =4; h_population[13][4] =9; h_population[13][5] =0; h_population[13][6] =1; h_population[13][7] =8; h_population[13][8] =5; h_population[13][9] =7;
	h_population[14][0] =4; h_population[14][1] =9; h_population[14][2] =5; h_population[14][3] =7; h_population[14][4] =2; h_population[14][5] =6; h_population[14][6] =0; h_population[14][7] =8; h_population[14][8] =1; h_population[14][9] =3;
	h_population[15][0] =6; h_population[15][1] =0; h_population[15][2] =9; h_population[15][3] =8; h_population[15][4] =2; h_population[15][5] =4; h_population[15][6] =7; h_population[15][7] =3; h_population[15][8] =1; h_population[15][9] =5;

	// Set Initial population in device memory from host memory with a fixed solution
	hipMemcpy(d_population, h_population,
			NSGA2_POPULATION_SIZE * (FACILITIES_LOCATIONS) * sizeof(short),
				hipMemcpyHostToDevice);

	printf("\nInitial Population\n");
	for (int i = 0; i < NSGA2_POPULATION_SIZE; i++) {
		printf("Chromosome %d\n", i);
		// Print solution.
		for (int j = 0; j < FACILITIES_LOCATIONS; j++) {
			printf("%d ", h_population[i][j]);
		}
		// Print fitness.
		for (int j = 0; j < OBJECTIVES; j++) {
			printf("|%d ", h_population_fitness[i][j]);
		}
		// Print rank.
		printf("|%d ", h_population_rank[i]);
		// Print crowding.
		printf("%f", h_population_crowding[i]);
		printf("\n");
	}
	
	/******************************************************************/

	for (int iteration = 1; iteration <= ITERATIONS; iteration++) {

		/* Calculate fitness on each population chromosome */
		parallelPopulationFitnessCalculation(h_population, h_population_fitness, d_population, d_population_fitness, d_sorted_population_fitness);
		parallelNSGA2(h_population, h_population_fitness, h_population_total_dominance, h_population_rank, h_population_crowding, d_population, d_population_fitness, d_sorted_population_fitness, d_population_total_dominance, d_population_rank, d_population_crowding);

	}
	printf("\nfitness\n");
	for (int i = 0; i < NSGA2_POPULATION_SIZE; i++) {
		for (int j = 0; j < OBJECTIVES; j++) {
			printf("|%d ", h_population_fitness[i][j]);
		}
		printf("\n");
	}
	clock_t end = clock();
	double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;


	/* Uncomment this section of code to print the Shuffled population
	Set current population (with Shuffled genes) in host memory from device memory */
	hipMemcpy(h_population, d_population,
		NSGA2_POPULATION_SIZE * FACILITIES_LOCATIONS * sizeof(short),
		hipMemcpyDeviceToHost);
	hipMemcpy(h_population_fitness, d_population_fitness,
		NSGA2_POPULATION_SIZE * (OBJECTIVES + 1) * sizeof(unsigned int),
		hipMemcpyDeviceToHost);
	hipMemcpy(h_population_rank, d_population_rank,
		NSGA2_POPULATION_SIZE * sizeof(short),
		hipMemcpyDeviceToHost);
	hipMemcpy(h_population_crowding, d_population_crowding,
		NSGA2_POPULATION_SIZE * sizeof(float),
		hipMemcpyDeviceToHost);

	printf("\nPopulation with fitness\n");
	for (int i = 0; i < NSGA2_POPULATION_SIZE; i++) {
		printf("Chromosome %d\n", i);
		// Print solution.
		for (int j = 0; j < FACILITIES_LOCATIONS; j++) {
			printf("%d ", h_population[i][j]);
		}
		// Print fitness.
		for (int j = 0; j < OBJECTIVES; j++) {
			printf("|%d ", h_population_fitness[i][j]);
		}
		// Print rank.
		printf("|%d ", h_population_rank[i]);
		// Print crowding.
		printf("%f", h_population_crowding[i]);
		printf("\n");
	}
	/* */

	printf("\n Time Spent: %f", time_spent);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

