#include "hip/hip_runtime.h"
/*
 * kenel.cu
 *
 *  Created on: May 19, 2019
 *      Author: Andrés Pupiales Arévalo
 *      apupiales@gmail.com
 *      https://github.com/apupiales
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation; either version 2 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * The GNU General Public License is available at:
 *   http://www.gnu.org/copyleft/gpl.html
 * or by writing to:
 *     The Free Software Foundation, Inc.,
 *     675 Mass Ave, Cambridge, MA 02139, USA.
 */

// System includes
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

//Custom sources
#include "settings_KC10_2fl_1uni.cu"
//#include "settings_KC10_2fl_1uni.cu"
//#include "settings_KC10_2fl_2uni.cu"
//#include "settings_KC10_2fl_3uni.cu"
//#include "settings_KC20_2fl_1uni.cu"
//#include "settings_KC20_2fl_2uni.cu"
//#include "settings_KC20_2fl_3uni.cu"
//#include "settings_KC30_2fl_1uni.cu"
//#include "settings_KC30_2fl_2uni.cu"

__global__ void curand_setup(hiprandState *state, int seed) {

	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	hiprand_init(seed, idx, 0, &state[idx]);
}

/**
 * This function creates the base population with POPULATION_SIZE chromosomes
 */
__global__ void generateBasePopulation(
		int population[][FACILITIES_LOCATIONS + OBJECTIVES]) {

	int i = blockIdx.x;
	int j = threadIdx.x;

	if (i < POPULATION_SIZE && j < FACILITIES_LOCATIONS + OBJECTIVES) {
		if (j < FACILITIES_LOCATIONS) {
			population[i][j] = j;
		} else {
			/* This positions will be use to allocate the fitness value
			 * for each objective and is initialized with 0 */
			population[i][j] = 0;
		}
	}
}

/**
 * This function shuffles chromosomes genes randomly over all population
 */
__global__ void shufflePopulationGenes(hiprandState *my_curandstate,
		const unsigned *max_rand_int, const unsigned *min_rand_int,
		int population[][FACILITIES_LOCATIONS + OBJECTIVES]) {

	int i = blockIdx.x;
	if (i < POPULATION_SIZE) {
		for (int j = 0; j < FACILITIES_LOCATIONS; j++) {
			int idx = j + blockDim.x * blockIdx.x;

			float myrandf = hiprand_uniform(my_curandstate + idx);
			int myrand = int(myrandf * 10);

			if (myrand != population[i][j]) {
				int current_value = population[i][j];
				population[i][j] = population[i][myrand];
				population[i][myrand] = current_value;
			}
		}
	}
}

/**
 * This function creates a base population binary 2d matrix representation.
 * population_2d variable will have the original binary 2d matrix representation
 * of each chromosome and population_2d_transposed will have the transposed version
 * of each binary 2d matrix representation.
 */
__global__ void populationTo2DRepresentation(
		int population[][FACILITIES_LOCATIONS + OBJECTIVES],
		int population_2d[][FACILITIES_LOCATIONS],
		int population_2d_transposed[][FACILITIES_LOCATIONS]) {

	int i = blockIdx.x;
	int j = threadIdx.x;
	int k = threadIdx.y;

	if (i < POPULATION_SIZE && j < FACILITIES_LOCATIONS
			&& k < FACILITIES_LOCATIONS) {
		if (population[i][j] == k) {
			population_2d[j + (i * FACILITIES_LOCATIONS)][k] = 1;
			population_2d_transposed[k + (i * FACILITIES_LOCATIONS)][j] = 1;
		} else {
			population_2d[j + (i * FACILITIES_LOCATIONS)][k] = 0;
			population_2d_transposed[k + (i * FACILITIES_LOCATIONS)][j] = 0;
		}
	}
}

/**
 *  Multiplication between selected flow matrix and input_matrix
 *  (in this strict order)
 */
__global__ void multiplicationWithFlowMatrix(int flow_matrix_id,
		int input_matrix[][FACILITIES_LOCATIONS],
		int output_matrix[][FACILITIES_LOCATIONS]) {

	int i = blockIdx.x;
	int j = threadIdx.x;
	int k = threadIdx.y;

	if (i < POPULATION_SIZE && j < FACILITIES_LOCATIONS
			&& k < FACILITIES_LOCATIONS) {
		int sum = 0;
		for (int x = 0; x < FACILITIES_LOCATIONS; x++) {
			sum += d_flowMatrices[flow_matrix_id][j][x]
					* input_matrix[x + (i * FACILITIES_LOCATIONS)][k];
		}
		output_matrix[j + (i * FACILITIES_LOCATIONS)][k] = sum;
	}
}

/**
 *  Multiplication between Transposed distance matrix and input_matrix
 *  (in this strict order)
 */
__global__ void multiplicationWithTranposedDistanceMatrix(
		int input_matrix[][FACILITIES_LOCATIONS],
		int output_matrix[][FACILITIES_LOCATIONS]) {

	int i = blockIdx.x;
	int j = threadIdx.x;
	int k = threadIdx.y;

	if (i < POPULATION_SIZE && j < FACILITIES_LOCATIONS
			&& k < FACILITIES_LOCATIONS) {
		int sum = 0;

		for (int x = 0; x < FACILITIES_LOCATIONS; x++) {
			sum += input_matrix[j + (i * FACILITIES_LOCATIONS)][x]
					* d_transposeDistancesMatrix[x][k];
		}
		output_matrix[j + (i * FACILITIES_LOCATIONS)][k] = sum;
	}
}

/**
 *  Multiplication between matrix a and matrix b
 */
__global__ void matrixMultiplication(int input_matrix_a[][FACILITIES_LOCATIONS],
		int input_matrix_b[][FACILITIES_LOCATIONS],
		int output_matrix[][FACILITIES_LOCATIONS]) {

	int i = blockIdx.x;
	int j = threadIdx.x;
	int k = threadIdx.y;

	if (i < POPULATION_SIZE && j < FACILITIES_LOCATIONS
			&& k < FACILITIES_LOCATIONS) {
		int sum = 0;
		for (int x = 0; x < FACILITIES_LOCATIONS; x++) {
			sum += input_matrix_a[j + (i * FACILITIES_LOCATIONS)][x]
					* input_matrix_b[x + (i * FACILITIES_LOCATIONS)][k];
		}
		output_matrix[j + (i * FACILITIES_LOCATIONS)][k] = sum;
	}
}

__global__ void calculateTrace(int objective_id,
		int input_matrix[][FACILITIES_LOCATIONS],
		int population[][FACILITIES_LOCATIONS + OBJECTIVES]) {

	int i = blockIdx.x;

	if (i < POPULATION_SIZE) {
		int sum = 0;
		for (int x = 0; x < FACILITIES_LOCATIONS; x++) {
			sum += input_matrix[x + (i * FACILITIES_LOCATIONS)][x];
		}
		population[i][FACILITIES_LOCATIONS + objective_id] = sum;
	}
}

/**
 * This function calculates the fitness of all chromosomes in the population.
 * The flow matrix is multiplied with the chromosome (represented in a binary
 * 2d matrix), the resultant matrix is multiplied with the distance transposed
 * matrix, then the resultant matrix is multiplied with the transposed chromosome
 * (also a binary 2d matrix). The trace of this resultant matrix is the chromosome's
 * fitness. The fitness must be calculated for each flow matrix.
 * Trace(Fn*X*DT*XT)
 */
void calculatePopulationfitness(
		int h_population[][FACILITIES_LOCATIONS + OBJECTIVES], int d_population[][FACILITIES_LOCATIONS + OBJECTIVES]) {

	/* Variable to check correct synchronization */
	hipError_t cudaStatus;

	/*******************************************************************************************
	 * Comment this section if you don't need to print the partial results
	 * of fitness calculation.
	 */

	/* Variable for population binary 2d representation in host memory (X)*/
	int h_2d_population[POPULATION_SIZE * FACILITIES_LOCATIONS][FACILITIES_LOCATIONS];

	/* Variable for population binary 2d representation transposed in host memory (XT) */
	int h_2d_transposed_population[POPULATION_SIZE * FACILITIES_LOCATIONS][FACILITIES_LOCATIONS];

	/* Variable to keep F1*X result in host memory (F1: Flow matrix 1)*/
	int h_temporal_1[POPULATION_SIZE * FACILITIES_LOCATIONS][FACILITIES_LOCATIONS];

	/* Variable to keep F2*X result in host memory (F2: Flow matrix 2)*/
	int h_temporal_2[POPULATION_SIZE * FACILITIES_LOCATIONS][FACILITIES_LOCATIONS];

	/* Variable to keep F1*X*DT result in host memory (DT: Transposed Distances matrix) */
	int h_temporal_3[POPULATION_SIZE * FACILITIES_LOCATIONS][FACILITIES_LOCATIONS];

	/* Variable to keep F2*X*DT result in host memory (DT: Transposed Distances matrix) */
	int h_temporal_4[POPULATION_SIZE * FACILITIES_LOCATIONS][FACILITIES_LOCATIONS];

	/* Variable to keep F1*X*DT*XT result in host memory*/
	int h_temporal_5[POPULATION_SIZE * FACILITIES_LOCATIONS][FACILITIES_LOCATIONS];

	/* Variable to keep F2*X*DT*XT result in host memory*/
	int h_temporal_6[POPULATION_SIZE * FACILITIES_LOCATIONS][FACILITIES_LOCATIONS];

	/********************************************************************************************/

	/* Variable for population binary 2d representation in device memory (X)*/
	int (*d_2d_population)[FACILITIES_LOCATIONS];
	hipMalloc((void**) &d_2d_population,
			sizeof(int) * POPULATION_SIZE * FACILITIES_LOCATIONS
					* FACILITIES_LOCATIONS);

	/* Variable for population binary 2d representation transposed in device memory (XT) */
	int (*d_2d_transposed_population)[FACILITIES_LOCATIONS];
	hipMalloc((void**) &d_2d_transposed_population,
			sizeof(int) * POPULATION_SIZE * FACILITIES_LOCATIONS
					* FACILITIES_LOCATIONS);

	/*
	 * Variable to keep F1*X result in device memory (F1: Flow matrix 1).
	 * This variable is also use to keep F1*X*DT*XT result
	 */
	int (*d_temporal_1)[FACILITIES_LOCATIONS];
	hipMalloc((void**) &d_temporal_1,
			sizeof(int) * POPULATION_SIZE * FACILITIES_LOCATIONS
					* FACILITIES_LOCATIONS);
	/*
	 * Variable to keep F1*X result in device memory (F2: Flow matrix 2).
	 * This variable is also use to keep F2*X*DT*XT result
	 */
	int (*d_temporal_2)[FACILITIES_LOCATIONS];
	hipMalloc((void**) &d_temporal_2,
			sizeof(int) * POPULATION_SIZE * FACILITIES_LOCATIONS
					* FACILITIES_LOCATIONS);
	/* Variable to keep F1*X*DT result in device memory (DT: Transposed Distances matrix) */
	int (*d_temporal_3)[FACILITIES_LOCATIONS];
	hipMalloc((void**) &d_temporal_3,
			sizeof(int) * POPULATION_SIZE * FACILITIES_LOCATIONS
					* FACILITIES_LOCATIONS);
	/* Variable to keep F2*X*DT result in device memory (DT: Transposed Distances matrix) */
	int (*d_temporal_4)[FACILITIES_LOCATIONS];
	hipMalloc((void**) &d_temporal_4,
			sizeof(int) * POPULATION_SIZE * FACILITIES_LOCATIONS
					* FACILITIES_LOCATIONS);

	dim3 threads(32, 32);
	populationTo2DRepresentation<<<POPULATION_SIZE, threads>>>(d_population,
			d_2d_population, d_2d_transposed_population);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Sync CudaError!");
	}

	/*********************************************************************
	 * Comment this section if you don't need to print the partial results
	 * of fitness calculation.
	 */

	/* Set current population binary 2d representation in host memory from device memory */
	hipMemcpy(h_2d_population, d_2d_population,
	POPULATION_SIZE * FACILITIES_LOCATIONS * FACILITIES_LOCATIONS * sizeof(int),
			hipMemcpyDeviceToHost);
	/* Set current population binary 2d representation transposed in host memory from device memory */
	hipMemcpy(h_2d_transposed_population, d_2d_transposed_population,
	POPULATION_SIZE * FACILITIES_LOCATIONS * FACILITIES_LOCATIONS * sizeof(int),
			hipMemcpyDeviceToHost);
	/*********************************************************************/

	/*
	 * F1*X
	 */
	multiplicationWithFlowMatrix<<<POPULATION_SIZE, threads>>>(0,
			d_2d_population, d_temporal_1);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Sync CudaError M1!");
	}
	/*
	 * F2*X
	 */
	multiplicationWithFlowMatrix<<<POPULATION_SIZE, threads>>>(1,
			d_2d_population, d_temporal_2);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Sync CudaError M2!");
	}

	/*********************************************************************
	 * Comment this section if you don't need to print the partial results
	 * of fitness calculation.
	 */

	/* Set the result of F1*X in host memory from device memory */
	hipMemcpy(h_temporal_1, d_temporal_1,
			POPULATION_SIZE * FACILITIES_LOCATIONS * FACILITIES_LOCATIONS
					* sizeof(int), hipMemcpyDeviceToHost);
	/* Set the result of F2*X in host memory from device memory */
	hipMemcpy(h_temporal_2, d_temporal_2,
			POPULATION_SIZE * FACILITIES_LOCATIONS * FACILITIES_LOCATIONS
					* sizeof(int), hipMemcpyDeviceToHost);
	/*********************************************************************/

	/*
	 * F1*X*DT
	 */
	multiplicationWithTranposedDistanceMatrix<<<POPULATION_SIZE, threads>>>(
			d_temporal_1, d_temporal_3);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Sync CudaError M3!");
	}
	/*
	 * F2*X*DT
	 */
	multiplicationWithTranposedDistanceMatrix<<<POPULATION_SIZE, threads>>>(
			d_temporal_2, d_temporal_4);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Sync CudaError M4!");
	}

	/*********************************************************************
	 * Comment this section if you don't need to print the partial results
	 * of fitness calculation.
	 */

	/* Set the result of F1*X*DT in host memory from device memory */
	hipMemcpy(h_temporal_3, d_temporal_3,
	POPULATION_SIZE * FACILITIES_LOCATIONS * FACILITIES_LOCATIONS * sizeof(int),
			hipMemcpyDeviceToHost);
	/* Set the result of F2*X*DT in host memory from device memory */
	hipMemcpy(h_temporal_4, d_temporal_4,
	POPULATION_SIZE * FACILITIES_LOCATIONS * FACILITIES_LOCATIONS * sizeof(int),
			hipMemcpyDeviceToHost);
	/*********************************************************************/

	/*
	 * F1*X*DT*XT
	 */
	matrixMultiplication<<<POPULATION_SIZE, threads>>>(d_temporal_3,
			d_2d_transposed_population, d_temporal_1);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Sync CudaError M5!");
	}
	/*
	 * F2*X*DT*XT
	 */
	matrixMultiplication<<<POPULATION_SIZE, threads>>>(d_temporal_4,
			d_2d_transposed_population, d_temporal_2);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Sync CudaError M6!");
	}

	/*********************************************************************
	 * Comment this section if you don't need to print the partial results
	 * of fitness calculation.
	 */

	/* Set the result of F1*X*DT*XT in host memory from device memory */
	hipMemcpy(h_temporal_5, d_temporal_1,
	POPULATION_SIZE * FACILITIES_LOCATIONS * FACILITIES_LOCATIONS * sizeof(int),
			hipMemcpyDeviceToHost);
	/* Set the result of F2*X*DT*XT in host memory from device memory */
	hipMemcpy(h_temporal_6, d_temporal_1,
	POPULATION_SIZE * FACILITIES_LOCATIONS * FACILITIES_LOCATIONS * sizeof(int),
			hipMemcpyDeviceToHost);
	/*********************************************************************/

	/*
	 * Trace(F1*X*DT*XT)
	 */
	calculateTrace<<<POPULATION_SIZE, 1>>>(0, d_temporal_1, d_population);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Sync CudaError M7!");
	}
	/*
	 * Trace(F2*X*DT*XT)
	 */
	calculateTrace<<<POPULATION_SIZE, 1>>>(1, d_temporal_2, d_population);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Sync CudaError M8!");
	}

	/* Set current population (with Shuffled genes) in host memory from device memory */
	hipMemcpy(h_population, d_population,
	POPULATION_SIZE * (FACILITIES_LOCATIONS + OBJECTIVES) * sizeof(int),
			hipMemcpyDeviceToHost);


	/*********************************************************************
	 * Comment this section if you don't need to print the partial results
	 * of fitness calculation.
	 * NOTE: If you uncomment this section, all previous sections with this
	 * notes must be uncomented too.
	 */
	printf("\nPopulation\n");
	for (int i = 0; i < POPULATION_SIZE; i++) {
		printf("Chromosome %d\n", i);
		for (int j = 0; j < FACILITIES_LOCATIONS + OBJECTIVES; j++) {
			printf("%d ", h_population[i][j]);
		}
		printf("\n");

		printf("\n2d Matrix Representation (X) \n");
		for (int x = 0; x < FACILITIES_LOCATIONS; x++) {

			for (int y = 0; y < FACILITIES_LOCATIONS; y++) {
				printf("%d ", h_2d_population[x + (i * FACILITIES_LOCATIONS)][y]);
			}
			printf("\n");
		}

		printf("\n2d Matrix Representation (Transposed) (XT)\n");
		for (int x = 0; x < FACILITIES_LOCATIONS; x++) {

			for (int y = 0; y < FACILITIES_LOCATIONS; y++) {
				printf("%d ", h_2d_transposed_population[x + (i * FACILITIES_LOCATIONS)][y]);
			}
			printf("\n");
		}

		printf("\n F1*X \n");
		for (int x = 0; x < FACILITIES_LOCATIONS; x++) {

			for (int y = 0; y < FACILITIES_LOCATIONS; y++) {
				printf("%d ", h_temporal_1[x + (i * FACILITIES_LOCATIONS)][y]);
			}
			printf("\n");
		}

		printf("\n F1*X*DT \n");
		for (int x = 0; x < FACILITIES_LOCATIONS; x++) {
			for (int y = 0; y < FACILITIES_LOCATIONS; y++) {
				printf("%d ", h_temporal_3[x + (i * FACILITIES_LOCATIONS)][y]);
			}
			printf("\n");
		}

		printf("\n F1*X*DT*XT \n");
		for (int x = 0; x < FACILITIES_LOCATIONS; x++) {
			for (int y = 0; y < FACILITIES_LOCATIONS; y++) {
				printf("%d ", h_temporal_5[x + (i * FACILITIES_LOCATIONS)][y]);
			}
			printf("\n");
		}

		printf("\n F2*X \n");
		for (int x = 0; x < FACILITIES_LOCATIONS; x++) {
			for (int y = 0; y < FACILITIES_LOCATIONS; y++) {
				printf("%d ", h_temporal_2[x + (i * FACILITIES_LOCATIONS)][y]);
			}
			printf("\n");
		}

		printf("\n F2*X*DT \n");
		for (int x = 0; x < FACILITIES_LOCATIONS; x++) {
			for (int y = 0; y < FACILITIES_LOCATIONS; y++) {
				printf("%d ", h_temporal_4[x + (i * FACILITIES_LOCATIONS)][y]);
			}
			printf("\n");
		}

		printf("\n F2*X*DT*XT \n");
		for (int x = 0; x < FACILITIES_LOCATIONS; x++) {
			for (int y = 0; y < FACILITIES_LOCATIONS; y++) {
				printf("%d ", h_temporal_6[x + (i * FACILITIES_LOCATIONS)][y]);
			}
			printf("\n");
		}
		printf("\n");
	}
	/*********************************************************************/

	hipFree(d_2d_population);
	hipFree(d_2d_transposed_population);
	hipFree(d_temporal_1);
	hipFree(d_temporal_2);
	hipFree(d_temporal_3);
	hipFree(d_temporal_4);
}

int main() {
	/* To measure the execution time */
	clock_t begin = clock();
	/* To set seed variable */
	time_t t;
	/* To check correct synchronization */
	hipError_t cudaStatus;

	/* Initializes random number generator */
	srand((unsigned) time(&t));

	/* seed for hiprand */
	int seed = rand() % 10000;

	/* Variable for population in host memory */
	int h_population[POPULATION_SIZE][FACILITIES_LOCATIONS + OBJECTIVES];

	/* Variable for population in device memory */
	int (*d_population)[FACILITIES_LOCATIONS + OBJECTIVES];
	hipMalloc((void**) &d_population,
			sizeof(int) * POPULATION_SIZE
					* (FACILITIES_LOCATIONS + OBJECTIVES));

	/* Generation of all base chromosomes (genes ordered ascending) */
	generateBasePopulation<<<POPULATION_SIZE, 32>>>(d_population);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Sync CudaError!");
	}
	/* Set population in host memory from device memory */
	hipMemcpy(h_population, d_population,
	POPULATION_SIZE * (FACILITIES_LOCATIONS + OBJECTIVES) * sizeof(int),
			hipMemcpyDeviceToHost);

	/* Uncommet this section of code to print the base population
	 printf("\nBase Population\n");
	 for (int i = 0; i < POPULATION_SIZE; i++) {
	 printf("Chromosome %d\n", i);
	 for (int j = 0; j < FACILITIES_LOCATIONS + OBJECTIVES; j++) {
	 printf("%d ", h_population[i][j]);
	 }
	 printf("\n");
	 }
	 */

	/* Initialize variables for random values generation with hiprand */
	hiprandState *d_state;
	hipMalloc(&d_state, sizeof(hiprandState));
	unsigned *d_max_rand_int, *h_max_rand_int, *d_min_rand_int, *h_min_rand_int;
	hipMalloc(&d_max_rand_int, sizeof(unsigned));
	h_max_rand_int = (unsigned *) malloc(sizeof(unsigned));
	hipMalloc(&d_min_rand_int, sizeof(unsigned));
	h_min_rand_int = (unsigned *) malloc(sizeof(unsigned));
	*h_max_rand_int = FACILITIES_LOCATIONS;
	*h_min_rand_int = 0;
	hipMemcpy(d_max_rand_int, h_max_rand_int, sizeof(unsigned),
			hipMemcpyHostToDevice);
	hipMemcpy(d_min_rand_int, h_min_rand_int, sizeof(unsigned),
			hipMemcpyHostToDevice);

	curand_setup<<<POPULATION_SIZE, 32>>>(d_state, seed);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Sync CudaError!");
	}

	/* Shuffles chromosome genes randomly over all population */
	shufflePopulationGenes<<<POPULATION_SIZE, 1>>>(d_state, d_max_rand_int,
			d_min_rand_int, d_population);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Sync CudaError!");
	}

	/* Set current population (with Shuffled genes) in host memory from device memory */
	hipMemcpy(h_population, d_population,
	POPULATION_SIZE * (FACILITIES_LOCATIONS + OBJECTIVES) * sizeof(int),
			hipMemcpyDeviceToHost);

	/* Uncommet this section of code to print the Shuffled population
	 printf("\nShuffled Population\n");
	 for (int i = 0; i < POPULATION_SIZE; i++) {
	 printf("Chromosome %d\n", i);
	 for (int j = 0; j < FACILITIES_LOCATIONS + OBJECTIVES; j++) {
	 printf("%d ", h_population[i][j]);
	 }
	 printf("\n");
	 }
	 */

	/* Set all chromosomes with 1 2 7 9 6 5 0 4 3 8 0 0 for test purposes*/
	for (int a = 0; a < POPULATION_SIZE; a++) {
		h_population[a][0] = 1;
		h_population[a][1] = 2;
		h_population[a][2] = 7;
		h_population[a][3] = 9;
		h_population[a][4] = 6;
		h_population[a][5] = 5;
		h_population[a][6] = 0;
		h_population[a][7] = 4;
		h_population[a][8] = 3;
		h_population[a][9] = 8;
		h_population[a][10] = 0;
		h_population[a][11] = 0;
	}

	hipMemcpy(d_population, h_population,
			POPULATION_SIZE * (FACILITIES_LOCATIONS + OBJECTIVES) * sizeof(int),
				hipMemcpyHostToDevice);

	/* Calculate fitness on each population chromosome */
	calculatePopulationfitness(h_population, d_population);

	clock_t end = clock();
	double time_spent = (double) (end - begin) / CLOCKS_PER_SEC;
	printf("\n Time Spent: %f", time_spent);

	exit(0);
}

